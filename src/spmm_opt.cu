#include "hip/hip_runtime.h"
#include "spmm_opt.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <string.h>
const int BLOCK_X = 32;
const int BLOCK_Y = 32;
const int NUM_THREADS = BLOCK_X * BLOCK_Y;

inline int ceil_div(int a, int b)
{
    return (a + b - 1) / b;
}

__global__ void spmm_kernel_notopt(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int feat_in)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_v)
        return;
    int begin = ptr[tid], end = ptr[tid + 1];
    for (int j = 0; j < feat_in; ++j)
    {
        float result = 0.0f;
        for (int i = begin; i < end; ++i)
        {
            // Transposing the vin maybe cache-friendly
            result += vin[idx[i] * feat_in + j] * val[i];
        }
        vout[tid * feat_in + j] = result;
    }
}

__global__ void spmm_kernel_merge(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int feat_in)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    // if (tid == 0) {
    //     printf("GridDim = <%d, %d, %d>\n", gridDim.x, gridDim.y, gridDim.z);
    //     printf("BlockDim = <%d, %d, %d>\n", blockDim.x, blockDim.y, blockDim.z);
    // }
    int x = tid / BLOCK_Y;
    if (x >= num_v)
        return;
    int line_id = threadIdx.x / BLOCK_Y;
    int lane_id = tid & (BLOCK_Y - 1);
    int y = blockIdx.y * BLOCK_Y + lane_id;
    int out_idx = x * feat_in + y;
    const float *vin_offset = vin + y;

    int begin = __ldg(ptr + x), end = __ldg(ptr + x + 1);
    int nnz = end - begin;
    float result = 0.f, v = 0.f;
    // float mul_temp[BLOCK_Y];
    __shared__ float val_temp[BLOCK_X][BLOCK_Y];
    __shared__ int col_temp[BLOCK_X][BLOCK_Y];

    int ii, col;
    for (int i = begin; i < end; i += BLOCK_Y)
    {
        ii = i + lane_id;
        if (ii < end)
        {
            col_temp[line_id][lane_id] = __ldg(idx + ii) * feat_in;
            val_temp[line_id][lane_id] = __ldg(val + ii);
            // ++val[ii];
        }
        else
        {
            col_temp[line_id][lane_id] = 0;
            val_temp[line_id][lane_id] = 0;
        }
        // __syncthreads();
#pragma unroll
        for (int j = 0; j < BLOCK_Y; ++j)
        {
            if (val_temp[line_id][j])
                result += val_temp[line_id][j] * __ldg(vin_offset + col_temp[line_id][j]);
        }
// #pragma unroll
//         for (int j = 0; j < BLOCK_X; ++j)
//         {
//             result += mul_temp[j];
//         }
    }
    vout[out_idx] = result;
}

void SpMMOpt::preprocess(float *vin, float *vout)
{
    // dbg("TODO");
    grid.x = ceil_div(num_v, BLOCK_X);
    grid.y = ceil_div(feat_in, BLOCK_Y);
    grid.z = 1;
    block.x = NUM_THREADS;
    block.y = 1;
    block.z = 1;
}

void SpMMOpt::run(float *vin, float *vout)
{
    // dbg("TODO");
    // spmm_kernel_opt<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
    // printf("num_v = %d, feat_in = %d\n", num_v, feat_in);
    // printf("Grid = <%d, %d, %d>\n", grid.x, grid.y, grid.z);
    // printf("Block = <%d, %d, %d>\n", block.x, block.y, block.z);
    // hipMemset(d_val, 0, sizeof(float) * num_e);
    spmm_kernel_merge<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
    // float *val = (float *)malloc(sizeof(float) * num_e);
    // hipMemcpy(val, d_val, num_e * sizeof(float), hipMemcpyDeviceToHost);
    // float sum = 0;
    // for (int i = 0; i < num_e; ++i)
    // {
    //     sum += val[i];
    // }
    // printf("Mean = %f\n", sum / num_e);
    // free(val);
}